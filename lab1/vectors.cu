#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define RANGE 19.87

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__
void vecGPU(float* ad, float* bd, float* cd, int n, int threadsPerBlock);

/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; //temp is just another copy of C
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	
	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device  
		*/

	    int size = n * sizeof(float);
		hipError_t err = hipSuccess;

	    err = hipMalloc((void **)&ad, size);
		if(err != hipSuccess)
		{
			fprintf(stderr, "Error allocating array ad on device: %s\n", hipGetErrorString(err));
			exit(1);
		}

		err = hipMalloc((void **)&bd, size);
		if(err != hipSuccess)
		{
			fprintf(stderr, "Error allocating array bd on device: %s\n", hipGetErrorString(err));
			exit(1);
		}

		err = hipMalloc((void **)&cd, size);
		if(err != hipSuccess)
		{
			fprintf(stderr, "Error allocating array cd on device: %s\n", hipGetErrorString(err));
			exit(1);
		}

		hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
		hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
		hipMemcpy(cd, c, size, hipMemcpyHostToDevice);
		
	/* TODO: 	
		3. write the kernel, call it: vecGPU
		4. call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to decide about the number of threads, blocks, etc and their geometry.
		*/
		int threadsPerBlock, blocksPerGrid;
		// 4 blocks and 500 threads per block
		blocksPerGrid = 4;
		threadsPerBlock = 500;
		// // 8 blocks and 500 threads per block
		// blocksPerGrid = 4;
		// threadsPerBlock = 500;
		// // 16 blocks and 500 blocks per block
		// blocksPerGrid = 4;
		// threadsPerBlock = 500;
		// // 4 blocks and 250 threads per block
		// blocksPerGrid = 4;
		// threadsPerBlock = 500;
		// // 8 blocks and 250 threads per block
		// blocksPerGrid = 4;
		// threadsPerBlock = 500;
		// // 16 blocks and 250 blocks per block
		// blocksPerGrid = 4;
		// threadsPerBlock = 500;

		// dim3 block(threadsPerBlock);
		// dim3 grid(blocksPerGrid);
		vecGPU<<<blocksPerGrid,threadsPerBlock>>>(ad, bd, cd, n, blocksPerGrid*threadsPerBlock);
		end = clock();
	/* TODO: 
		5. bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/
        hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);

		hipFree(ad);
		hipFree(bd);
		hipFree(cd);
	
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	int wrongCount = 0;
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++)
	{
		if( fabs(temp[i] - c[i]) >= 0.009) //compare up to the second degit in floating point
		{
			printf("Element %d in the result array does not match the sequential version\n", i);
			// printf("%d:\ta %.2lf\tb %.2lf\tc\tsequential %.2lf\tGPU %.2lf\n\n", i, a[i], b[i], temp[i], c[i]);
			wrongCount++;
		}
	}

	printf("Number of c values:\twrong %d\tcorrect %d\n", wrongCount, n-wrongCount);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/

__global__
void vecGPU(float* ad, float* bd, float* cd, int n, int totalNumOfThreads)
{
    // int totalNumOfThreads = blockDim.x * gridDim.x;
	int i, j, index;
	j = threadIdx.x + blockDim.x * blockIdx.x;
	for (i = 1; i <= (n/totalNumOfThreads + 1); i++)
	{
		index = i * j;
		if(index<n)
		{
			cd[index] += ad[index] * bd[index];
		}
	}
}

